
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// NOTE: Currently only arithmatic operations (+, -, /, *, ^) are supported

#define BLOCK_SIZE 1024
#define CEIL(a, b) ((a-1)/b +1)
#define BUFFER 256 // max number of characters in variables and membrane names

#define END 0
#define CON -1
#define VAR -2

// operators in order of precedence
#define LB  1   // (
#define EQ  2   // ==
#define NE  3   // !=
#define LT  4   // <
#define LE  5   // <=
#define GT  6   // >
#define GE  7   // >=
#define ADD 8   // +
#define SUB 9   // -
#define MUL 10  // *
#define MOD 11  // %
#define DIV 12  // /
#define EXP 13  // ^
#define NEG 14  // !
#define RB  15  // )

// limit for checking equality of 2 float values
#define LIMIT (float)0.00001

// NOTE: The cumulative length of production and distribution shouldn't be greater than INT_MAX

// check error
#define printError(func)                                                \
{                                                                       \
  hipError_t E  = func;                                                \
  if(E != hipSuccess)                                                  \
  {                                                                     \
    printf( "\nError at line: %d ", __LINE__);                          \
    printf( "\nError:  %s ", hipGetErrorString(E));                    \
  }                                                                     \
} \

//for postfix expression
typedef struct {
    int type;
    int i, j;
    float value;

} PostfixElement;

typedef struct {
    int proportion; // is zero for eof
    int i, j; // position of variable in variables list
} DistFuncElement;

typedef struct {
    int i, j; // position of the enzyme in variables list
} EnzymeObject;

// return precedence of operators
__device__ int precedence(int op) {
    return op;
}

// used to parse constant from the input string, updates string iterator i
__device__ float parseConstant(const char exprParsed[], int *i, int end = INT_MAX) {
   
    float temp = 0;

    while(exprParsed[*i] >= '0' && exprParsed[*i] <= '9' && (*i) < end) {
        temp = temp*10 + (exprParsed[*i] - '0');
        (*i)++;

        if(exprParsed[*i] == '.') {
            (*i)++;
            float mul = 10;
            while(exprParsed[*i] >= '0' && exprParsed[*i] <= '9' && (*i) < end) {
                temp = temp + (exprParsed[*i] - '0')/mul;
                mul *= 10; 
                (*i)++;
            }
        }
    }

    return temp;
}

// used to parse variable from the input string, updates string iterator i
__device__ int* parseVarible(const char exprParsed[], int *i, int end = INT_MAX) {
    
    int* p = (int*) malloc(sizeof(int)*2);
    *i = *i+2;

    p[0] = parseConstant(exprParsed, i, end);
    (*i)++;
    p[1] = parseConstant(exprParsed, i, end);

    return p;
}

// used to parse operator from the input string, updates string iterator i
__device__ int parseOp(const char exprParsed[], int *i) {
    
    int temp = 0;

    switch(exprParsed[*i]) {
        case '+': 
            temp = ADD;
            break;
        case '-':
            temp = SUB;
            break;
        case '*':
            temp = MUL;
            break;
        case '/':
            temp = DIV;
            break;
        case '^':
            temp = EXP;
            break;
        case '%':
            temp = MOD;
            break;
        case '(':
            temp = LB;
            break;
        case ')':
            temp = RB;
            break;
        case '<':
            temp = LT; // <
            if(exprParsed[*i+1] == '=') { // <=
                temp = LE;
                (*i)++;
            }
            break;
        case '>':
            temp = GT; // >
            if(exprParsed[*i+1] == '=') { // >=
                temp = GE;
                (*i)++;
            }
            break;
        case '=':
            if(exprParsed[*i+1] == '=') { // ==
                temp = EQ;
                (*i)++;
            }
        case '!':
            temp = NEG; // !
            if(exprParsed[*i+1] == '=') { // !=
                temp = NE;
                (*i)++;
            }
            break;
    }

    (*i)++;
    return temp;
}

/* Kernel function :
*  Each thread is assigned a set of programs in the ENPS model. They perform the 
*  following tasks on their respective programs:
*  1. Parses production function string and converts it to postfix expression
*  2. Parses distribution function into easy to handle data structure 
*  3. Simulates program "steps" number of times
*/
__global__ void enps(char *prodFunction, int *posProd, char *distFunction, int *posDist, int *numberOfVariables, 
	float *variables, int steps, int numberOfPrograms, EnzymeObject *enzymes, PostfixElement *postfix, 
    DistFuncElement *distribution, int numberOfMembranes, int *stackOfOps, float *stackPostfixEval, 
    float *minVariableInPosFunc, float *valueOfProdFunc, float *sumOfProportions, bool *isProgramActive) {
    
    
    int ID = threadIdx.x; //index of each thread

    for(int id = ID; id < numberOfPrograms; id += blockDim.x) {
        /* 
        *  pbegin : beginnig of production function
        *  pend   : ending of production function
        *  dbegin : beginning of distibution function
        *  dend   : ending of distribution function
        *           of each thread's respective programs
        */

        int pbegin = posProd[id], pend = posProd[id+1], dbegin = posDist[id], dend = posDist[id+1];

        int top = -1; // top of stackOfOps
        int pos = 0; // pos of postfix expression
        int i;

        // converting production function to postfix expression

        for(i = pbegin; i < pend;) {
            if(prodFunction[i] == '$') { // parsing a variable
                int *p = parseVarible(prodFunction, &i, pend);
                postfix[pos+pbegin].type = VAR;
                postfix[pos+pbegin].i = p[0];
                postfix[pos+pbegin].j = p[1];
                pos++;
            }
            else if(prodFunction[i] >= '0' && prodFunction[i] <= '9') { // parsing a constant
                float constant = parseConstant(prodFunction, &i, pend);
                postfix[pos+pbegin].type = CON;
                postfix[pos+pbegin].value = constant;
                pos++;
            }
            else { // parsing an operator
                int op = parseOp(prodFunction, &i);  

                if(op == LB) { // is '('
                    //push to stack
                    top++;
                    stackOfOps[top + pbegin] = op;
                }
                else if(op == RB) { // is ')'
                    while(top != -1 && stackOfOps[top + pbegin] != LB) { //not '('
                        int temp = stackOfOps[top + pbegin];
                        top--;
                        postfix[pos+pbegin].type = temp;
                        pos++;
                    }
                    top--; // pop '('
                }
                else { // is operator
                    while(top != -1 && precedence(op) <= precedence(stackOfOps[top + pbegin])) {
                        int temp = stackOfOps[top + pbegin];
                        top--;
                        postfix[pos+pbegin].type = temp;
                        pos++;
                    }

                    // push to stack
                    top++;
                    stackOfOps[top + pbegin] = op;
                }
            }
        }

        // pop out any remaining elements in stackOfOps
        while(top != -1) {
            int temp = stackOfOps[top + pbegin];
            top--;
            postfix[pos+pbegin].type = temp;
            pos++;
        } 

        // add terminating character
        postfix[pos+pbegin].type = END;
        pos++; // (possible optimization, this anyways stores last index, eof not neccesary)

        int pos1 = 0; // position of distribution
        sumOfProportions[id] = 0; // sum of proportions of distribution function

        // extracting proportion and variable from distribution function 
        for(i = dbegin; i < dend; ) {

            distribution[pos1+dbegin].proportion = parseConstant(distFunction, &i, dend);
            i++;

            int *p = parseVarible(distFunction, &i, dend);

            distribution[pos1+dbegin].i = p[0];
            distribution[pos1+dbegin].j = p[1];

            sumOfProportions[id] += distribution[pos1+dbegin].proportion;

            pos1++;
            i++;

        }
        
        distribution[pos1+dbegin].proportion = 0; //terminating condition

    }

    __syncthreads();

    int step = 0;   
    
    // computation for given number of steps
    while(step < steps) {

        for(int id = ID; id < numberOfPrograms; id += blockDim.x) {

            int pbegin = posProd[id];

            int pos = 0;
            int top = -1;
            minVariableInPosFunc[id] = 1e300; // minimum value of variables in production function

            while(postfix[pos+pbegin].type != END) {
                
                if(postfix[pos+pbegin].type == CON) { // constant
                    top++;
                    stackPostfixEval[top + pbegin] = postfix[pos+pbegin].value;
                }
                else if(postfix[pos+pbegin].type == VAR) { // variable
                    int a = postfix[pos+pbegin].i-1, b = postfix[pos+pbegin].j-1;
                    top++;
                    stackPostfixEval[top + pbegin] =  variables[numberOfVariables[b] + a];
                    if(variables[numberOfVariables[b] + a] < minVariableInPosFunc[id])
                        minVariableInPosFunc[id] = variables[numberOfVariables[b] + a];
                }
                else { // operator
                    float a = stackPostfixEval[top + pbegin - 1], b = stackPostfixEval[top + pbegin], result = 0;
                    top--;
                    switch(postfix[pos+pbegin].type) {
                        case ADD:
                            result = a + b;
                            break;
                        case SUB:
                            result = a - b;
                            break;
                        case MUL:
                            result = a * b;
                            break;
                        case DIV:
                            result = a / b;
                            break;
                        case EXP:
                            result = powf(a, b);
                            break;
                        case MOD:
                            result = fmod(a, b);
                            break;
                        case LT:
                            result = (a < b)?1:0;
                            break;
                        case GT:
                            result = (a > b)?1:0;
                            break;
                        case LE:
                            result = (a <= b)?1:0;
                            break;
                        case GE:
                            result = (a >= b)?1:0;
                            break;
                        case EQ:
                            result = (fabs(a-b) < LIMIT)?1:0;
                            break;
                        case NE:
                            result = (a != b)?1:0;
                            break;
                        case NEG:
                            result = (!b);
                            break;
                    }
                    stackPostfixEval[top + pbegin] = result;
                }
                pos++;

            }

            if(minVariableInPosFunc[id] == 1e300) { // if no variable in production function
                minVariableInPosFunc[id] = -1;
            } 

            valueOfProdFunc[id] = stackPostfixEval[pbegin];
        }

        __syncthreads();

        for(int id = ID; id < numberOfPrograms; id += blockDim.x) {
         
            int a = enzymes[id].i - 1, b = enzymes[id].j - 1;
            isProgramActive[id] = (variables[numberOfVariables[b] + a] > minVariableInPosFunc[id] &&
                 variables[numberOfVariables[b] + a] > 0)?true:false; // for POS
        }

        __syncthreads();

        for(int id = ID; id < numberOfPrograms; id += blockDim.x) {

            int pbegin = posProd[id];

            if(isProgramActive[id]) {   // if production function is active
                int pos = 0;

                while(postfix[pos+pbegin].type != END) {
                    
                    if(postfix[pos+pbegin].type == VAR) {
                        int a = postfix[pos+pbegin].i-1, b = postfix[pos+pbegin].j-1;
                        variables[numberOfVariables[b] + a] = 0;
                    }
                    
                    pos++;
                }
            }
            
        }

        __syncthreads();

        for(int id = ID; id < numberOfPrograms; id += blockDim.x) {
        
            int dbegin = posDist[id];
            int pos1 = 0;
            // distribute among variables

            if(isProgramActive[id]) { // if production function is active
                while(distribution[pos1+dbegin].proportion != 0) { 
                    int a = distribution[pos1+dbegin].i - 1, b = distribution[pos1+dbegin].j - 1;
                    atomicAdd(variables + numberOfVariables[b] + a, valueOfProdFunc[id] * (((float)distribution[pos1+dbegin].proportion)/sumOfProportions[id]));
                    pos1++;
                }
            }
        }

        step++;
        __syncthreads();
    }
}


int main(int argc, char **argv) {

    FILE *ptr = fopen(argv[1], "r");
    
    int ctr;

    // Reading number of programs
    int numberOfPrograms;
    fscanf(ptr, "%d", &numberOfPrograms);

    // Reading number of membranes
    int numberOfMembranes;
    fscanf(ptr, "%d", &numberOfMembranes);

    // Reading the production functions
    int sizeOfProdFunction;
    fscanf(ptr, "%d", &sizeOfProdFunction);
    char *prodFunction = (char*)malloc(sizeOfProdFunction+1);
    fscanf(ptr, "%s", prodFunction);

    // Reading the position of production functions
    int sizeOfPosProd;
    fscanf(ptr, "%d", &sizeOfPosProd);
    int *posProd = (int*)malloc(sizeOfPosProd*sizeof(int));
    for(ctr=0; ctr<sizeOfPosProd; ctr++){
        fscanf(ptr, "%d", &posProd[ctr]);
    }


    // Reading the distribution functions
    int sizeOfDistFunction;
    fscanf(ptr, "%d", &sizeOfDistFunction);
    char *distFunction = (char*)malloc(sizeOfDistFunction+1);
    fscanf(ptr, "%s", distFunction);

    // Reading the position of distribution functions
    int sizeOfPosDist;
    fscanf(ptr, "%d", &sizeOfPosDist);
    int *posDist = (int*)malloc(sizeOfPosDist*sizeof(int));
    for(ctr=0; ctr<sizeOfPosDist; ctr++){
        fscanf(ptr, "%d", &posDist[ctr]);
    }

    // Reading the cumulative number of variables
    int sizeOfNumberOfVariables;
    fscanf(ptr, "%d", &sizeOfNumberOfVariables);
    int *numberOfVariables = (int*)malloc(sizeOfNumberOfVariables*sizeof(int));
    for(ctr=0; ctr<sizeOfNumberOfVariables; ctr++){
        fscanf(ptr, "%d", &numberOfVariables[ctr]);
    }
    int max_a = 1, max_b = ctr + 1; // programs with no enzymes are assigned with this enzyme indices 

    // Reading the variable values
    int sizeOfVariables;
    fscanf(ptr, "%d", &sizeOfVariables);
    sizeOfVariables++; // to accomadate very large enzyme (for programs without enzymes)
    float *variables = (float*)malloc(sizeOfVariables*sizeof(float));
    for(ctr=0; ctr<sizeOfVariables-1; ctr++){
        fscanf(ptr, "%f", &variables[ctr]);
    }
    variables[ctr] = 1e300; // programs with no enzyme is assigned this enzyme

    int sizeOfEnzymes = numberOfPrograms;
    EnzymeObject *enzymes = (EnzymeObject*)malloc(sizeOfEnzymes*sizeof(EnzymeObject)); // can directly assign size of programs
    for(ctr=0; ctr < sizeOfEnzymes; ctr++) {

        int a, b;
        fscanf(ptr, "%d", &a);
        fscanf(ptr, "%d", &b);

        if(a == -1 && b == -1){ // if no enzyme assigned to the program
            enzymes[ctr].i = max_a;
            enzymes[ctr].j = max_b;
        }
        else{
            enzymes[ctr].i = a;
            enzymes[ctr].j = b;
        }
    }



    // Reading the number of steps
    int numberOfIterations;
    fscanf(ptr, "%d", &numberOfIterations);

    char *d_prodFunction, *d_distFunction;
    int *d_posProd, *d_posDist, *d_numberOfVariables, *d_stackOfOps;
    float *d_variables, *d_stackPostfixEval, *d_minVariableInPosFunc, *d_valueOfProdFunc, *d_sumOfProportions;
    bool *d_isProgramActive;
    EnzymeObject *d_enzymes;
    PostfixElement *d_postfix;
    DistFuncElement* d_distribution;

    printf("Allocating memory\n");
    printError(hipMalloc((void**) &d_prodFunction, sizeOfProdFunction * sizeof(char)));
    printError(hipMalloc((void**) &d_posProd, sizeOfPosProd * sizeof(int)));
    printError(hipMalloc((void**) &d_distFunction, sizeOfDistFunction * sizeof(char)));
    printError(hipMalloc((void**) &d_posDist, sizeOfPosDist * sizeof(int)));
    printError(hipMalloc((void**) &d_numberOfVariables, sizeOfNumberOfVariables * sizeof(int)));
    printError(hipMalloc((void**) &d_variables, sizeOfVariables * sizeof(float)));
    printError(hipMalloc((void**) &d_enzymes, sizeOfEnzymes * sizeof(EnzymeObject)));
    printError(hipMalloc((void**) &d_postfix, sizeOfProdFunction * sizeof(PostfixElement)));
    printError(hipMalloc((void**) &d_distribution, sizeOfDistFunction * sizeof(DistFuncElement)));
    printError(hipMalloc((void**) &d_stackOfOps, sizeOfProdFunction * sizeof(int)));
    printError(hipMalloc((void**) &d_stackPostfixEval, sizeOfProdFunction * sizeof(float)));
    printError(hipMalloc((void**) &d_minVariableInPosFunc, numberOfPrograms * sizeof(float)));
    printError(hipMalloc((void**) &d_valueOfProdFunc, numberOfPrograms * sizeof(float)));
    printError(hipMalloc((void**) &d_sumOfProportions, numberOfPrograms * sizeof(float)));
    printError(hipMalloc((void**) &d_isProgramActive, numberOfPrograms * sizeof(bool)));
    printf("Allocated memory\n");
        
    printError(hipMemcpy(d_prodFunction, prodFunction, sizeOfProdFunction * sizeof(char), hipMemcpyHostToDevice));
    printError(hipMemcpy(d_posProd, posProd, sizeOfPosProd * sizeof(int), hipMemcpyHostToDevice));
    printError(hipMemcpy(d_distFunction, distFunction, sizeOfDistFunction * sizeof(char), hipMemcpyHostToDevice));
    printError(hipMemcpy(d_posDist, posDist, sizeOfPosDist * sizeof(int), hipMemcpyHostToDevice));
    printError(hipMemcpy(d_numberOfVariables, numberOfVariables, sizeOfNumberOfVariables * sizeof(int), hipMemcpyHostToDevice));
    printError(hipMemcpy(d_variables, variables, sizeOfVariables * sizeof(float), hipMemcpyHostToDevice));
    printError(hipMemcpy(d_enzymes, enzymes, sizeOfEnzymes * sizeof(EnzymeObject), hipMemcpyHostToDevice));
    printf("Copied data\n");

    // Kernel call

    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	printf("Entered Kernel\n");
    enps<<<1, BLOCK_SIZE>>>(d_prodFunction, d_posProd, d_distFunction, 
    	d_posDist, d_numberOfVariables, d_variables, numberOfIterations, numberOfPrograms, d_enzymes,  
        d_postfix, d_distribution, numberOfMembranes, d_stackOfOps, d_stackPostfixEval, d_minVariableInPosFunc, 
        d_valueOfProdFunc, d_sumOfProportions, d_isProgramActive);

    hipDeviceSynchronize();
    hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("Exited kernel\n");
	printf("Time taken : %f ms\n", milliseconds);
    
    hipMemcpy(variables, d_variables, sizeOfVariables * sizeof(float), hipMemcpyDeviceToHost);

    char name[BUFFER+1];
    printf("\nComputed variable values:\n\n");
    
    // Output computed values

    printf("num_ps = {\n");
    for(int i = 0; i < numberOfMembranes; i++) {

        fscanf(ptr, "%s", name);
        printf("  %s:\n", name);
        
        printf("    var = {");
        int j;
        for(j = numberOfVariables[i]; j < numberOfVariables[i+1]; j++) {
            fscanf(ptr, "%s", name);
            if(name[0] == '$') {
                break;
            }
    		printf(" %s: %f, ", name, variables[j]);
        } 

        printf("}\n    E = {");

        for(; j < numberOfVariables[i+1]; j++) {
            fscanf(ptr, "%s", name);
            printf(" %s: %f, ", name, variables[j]);
        }

        printf("}\n");

    }
    printf("}\n");

    return 0;
}
